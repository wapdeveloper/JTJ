#include "hip/hip_runtime.h"
#include<stdio.h>
#include < hip/hip_runtime.h>
#include <d:/book.h>
#define N   1000
#define M   128
#define K   2
#define txPerBlock  32


__global__ void jtj_cam( int * J,	int* blocks ,int* cameraID ,int *cameraCount)
{
	//���乲���ڴ�
	__shared__ int value[txPerBlock * 8]; 
	int index=blockIdx.x*blockDim.x;
	//������
	int colpos=  threadIdx.x & 0x7;      
	//������
	int rowpos = threadIdx.x - colpos; 
	int row[8] = {0, 0, 0, 0, 0, 0, 0, 0};
	//ȡ��ÿ�������Ӧ���Ÿ��Ⱦ���
	int tid =threadIdx.x+cameraID[blockIdx.x];
	while (tid < cameraCount[blockIdx.x]) 
	{
		//���Ÿ��Ⱦ������빲���ڴ�
		value[threadIdx.x] = J[tid];
		for(int j = 0; j < 8; ++j)   
		if(j>colpos||j==colpos) //���ٶԽ��ظ�����
			row[j] += (value[threadIdx.x] * value[rowpos + j]);
		tid += txPerBlock;
	}
	//�߳�ͬ��
	__syncthreads();
	//ÿ���̼߳������
	for(int i = 0; i < 8; ++i)
		value[threadIdx.x * 8 + i] = row[i]; 
	//�����ڴ��Լ
	int i=8*txPerBlock;
	while (i > 64) 
	{	
		tid =threadIdx.x;
		while(tid < (i>>1))
		{
			value[tid] += value[tid + (i>>1)];
			tid += txPerBlock;
		}  
		i >>=1;
	}
	//�õ�ÿ�������Ӧ�ľ���64��ֵ
	tid=threadIdx.x;
	while(tid<64)
	{
  		blocks[tid+(blockIdx.x<<6)]=value[tid];	
		tid+=txPerBlock;
	}
}

int main( void ) 
{
	int j[N], jtj[M],cameraID[K],cameraCount[K];
	int *dev_j,  *dev_jtj ,*dev_cameraID,*dev_cameraCount;

	//����GPU�Դ�
	HANDLE_ERROR( hipMalloc( (void**)&dev_j, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_jtj, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_cameraID, K * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_cameraCount, K * sizeof(int) ) );

	//��ȡ�ſɱȾ���
	const char *rpc1Filename = "in.txt";
	FILE* fid21 = fopen(rpc1Filename, "rt");
	for (int i = 0; i < N; i++)
	{
		fscanf(fid21, "%d ", &j[i]);
	}

	////��ȡ�ſɱȾ���
	//rpc1Filename = "D:\\f.txt";
	//fid21 = fopen(rpc1Filename, "rt");
	//for (int i = 0; i < N; i++)
	//{
	//	fscanf(fid21, "%d ", &jtj[i]);
	//}

	cameraID[0]=0;
	cameraID[1]=256;

	cameraCount[0]=256;
	cameraCount[1]=1000;


	// �ſɱȾ��󿽱���GPUx
	HANDLE_ERROR( hipMemcpy( dev_j, j, N * sizeof(int),
		hipMemcpyHostToDevice ) );	
	HANDLE_ERROR( hipMemcpy( dev_jtj, jtj, M * sizeof(int),
		hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_cameraID, cameraID, K * sizeof(int),
		hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_cameraCount, cameraCount, K * sizeof(int),
		hipMemcpyHostToDevice ) );

	//�˺���
	dim3 grid(2), block(txPerBlock);
	jtj_cam<<<grid, block>>>(dev_j, dev_jtj,dev_cameraID,dev_cameraCount);


	//��CPU������GPU

	HANDLE_ERROR( hipMemcpy( j, dev_j, N * sizeof(int),hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy( jtj, dev_jtj, M * sizeof(int),hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy( cameraID, dev_cameraID, K * sizeof(int),hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy(cameraCount, dev_cameraCount, K * sizeof(int),hipMemcpyDeviceToHost ) );


	// display the results
	for (int i=0; i<M; i++) 
	{
		printf( "%d   ", jtj[i] );
		if((i+1)%8==0)
			printf( "\n");
	}

	// free the memory allocated on the GPU
	HANDLE_ERROR( hipFree( dev_j ) );
	HANDLE_ERROR( hipFree( dev_jtj ) );
	HANDLE_ERROR( hipFree( dev_cameraID ) );
	HANDLE_ERROR( hipFree( dev_cameraCount ) );
	return 0;
}
