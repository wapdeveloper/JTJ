#include "hip/hip_runtime.h"
#include<stdio.h>
#include < hip/hip_runtime.h>
#include <d:/book.h>
#define N   1000
#define M   64
#define threadsPerBlock   64


__global__ void jtj_each_cam( float * J,	float* blocks ,int nJEC)
{
	__shared__ float value[threadsPerBlock * 8]; 
	int index=threadIdx.x + blockIdx.x * blockDim.x;
	int colpos=  threadIdx.x & 0x7;      
	int rowpos = threadIdx.x - colpos; 
	float row[8] = {0, 0, 0, 0, 0, 0, 0, 0};
	int tid =index;
	while (tid < nJEC) 
	{
		value[threadIdx.x] = J[tid];
		for(int j = 0; j < 8; ++j)   
		if(j>=colpos)
			row[j] += (value[threadIdx.x] * value[rowpos + j]);
		tid += gridDim.x*blockDim.x;	
	}
	for(int i = 0; i < 8; ++i)   
		value[threadIdx.x * 8 + i] = row[i]; 
	int i=8*threadsPerBlock;
	while (i != 64) 
	{	
		tid =threadIdx.x;
		while(tid < i/2)
		{
			value[tid] += value[tid + i/2];
			tid += threadsPerBlock;
		}  
		i /= 2;
	}
	blocks[index]=value[threadIdx.x];
    blocks[index]=blocks[index]+blocks[index+64];
}

int main( void ) 
{
	float j[N], jtj[M];
	float *dev_j,  *dev_jtj;

	//����GPU�Դ�
	HANDLE_ERROR( hipMalloc( (void**)&dev_j, N * sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_jtj, N * sizeof(float) ) );



	//��ȡ�ſɱȾ���
	const char *rpc1Filename = "D:\\in.txt";
	FILE* fid21 = fopen(rpc1Filename, "rt");
	for (int i = 0; i < N; i++)
	{
		fscanf(fid21, "%f ", &j[i]);
	}


	// �ſɱȾ��󿽱���GPU
	HANDLE_ERROR( hipMemcpy( dev_jtj, jtj, M * sizeof(float),
		hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_j, j, N * sizeof(float),
		hipMemcpyHostToDevice ) );

	//�˺���
	dim3 grid(2), block(threadsPerBlock);
	jtj_each_cam<<<grid, block>>>(
		dev_j, dev_jtj,N);


	//��CPU������GPU
	HANDLE_ERROR( hipMemcpy( jtj, dev_jtj, M * sizeof(float),hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy( j, dev_j, N * sizeof(float),hipMemcpyDeviceToHost ) );

	// display the results
	for (int i=0; i<M; i++) 
	{
		printf( "%d   ", jtj[i] );
		if((i+1)%8==0)
			printf( "\n");
	}

	// free the memory allocated on the GPU
	HANDLE_ERROR( hipFree( dev_j ) );
	HANDLE_ERROR( hipFree( dev_jtj ) );

	return 0;
}
