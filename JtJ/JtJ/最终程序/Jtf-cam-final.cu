#include "hip/hip_runtime.h"
texture<float4, 1, hipReadModeElementType> tex_jtc_cam;
texture<float, 1, hipReadModeElementType> tex_f_cam;

__global__ void jtf_cam(float* result, int* cameraID, int *cameraCount)
{
	//���乲���ڴ�
	__shared__ float value[txPerBlock * 8]; 
	float jx = 0, jy = 0, jz = 0, jw = 0; 
	//ȡ��ÿ��Jtc�����Լ���Ӧ��ͶӰ���
	int tid = threadIdx.x + (cameraID[blockIdx.x] >> 3);
	while (tid < cameraCount[blockIdx.x]>>3) 
	{
	    //ʰȡ��Jtc����������ڴ� 
	    float4 jab = tex1Dfetch(tex_jtc_cam, (tid * 2) + threadIdx.y);
		//ʰȡ��ͶӰ���f�������ڴ�
		float err = tex1Dfetch(tex_f_cam, tid);
		jx += jab.x * err;		jy += jab.y * err;
        jz += jab.z * err;  	jw += jab.w * err;
		tid += txPerBlock;
	}
	//�߳�ͬ��
	__syncthreads();
	//��ÿ���̼߳���õ���Jtf������빲���ڴ�
	int index=(threadIdx.x + (threadIdx.y * txPerBlock)) << 2;  
	value[index + 0] = jx;		value[index + 1] = jy;
	value[index + 2] = jz;		value[index + 3] = jw;
	//�����ڴ��Լ
	int i = txPerBlock * 8;
	while (i > 8) 
	{	
		tid = threadIdx.x + (threadIdx.y << n);
		while(tid < ((threadIdx.y << n) + (i >> 2)))
		{
			value[tid] += value[tid + (i >> 2)];
			tid += txPerBlock;
		}  
		i >>= 1;
	}
	//�õ�Jtf�����ÿ�������Ӧ8��ֵ
	if(threadIdx.x < 4)
        result[threadIdx.x + (threadIdx.y << 2) + (blockIdx.x << 3)]=value[threadIdx.x + (threadIdx.y << n)];
}


