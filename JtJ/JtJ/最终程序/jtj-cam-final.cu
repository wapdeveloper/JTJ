#include "hip/hip_runtime.h"

__global__ void jtj_cam(float* blocks, float * J, float* cameraID, float *cameraCount)
{
	//���乲���ڴ�
	__shared__ float value[txPerBlock * 8]; 
	int index = blockIdx.x * blockDim.x;
	//������
	int colpos = threadIdx.x & 0x7;      
	//������
	int rowpos = threadIdx.x - colpos; 
	int row[8] = {0, 0, 0, 0, 0, 0, 0, 0};
	//ȡ��Jc����Ԫ��
	int tid = threadIdx.x + cameraID[blockIdx.x];
	while (tid < cameraCount[blockIdx.x]) 
	{
		//��Jc�������빲���ڴ�
		value[threadIdx.x] = J[tid];
		for(int j = 0; j < 8; ++j)   
		if(j > colpos || j == colpos) //���ٶԽ��ظ�����
			row[j] += (value[threadIdx.x] * value[rowpos + j]);
		tid += txPerBlock;
	}
	//�߳�ͬ��
	__syncthreads();
	//ÿ���̼߳������
	for(int i = 0; i < 8; ++i)
		value[threadIdx.x * 8 + i] = row[i]; 
	//�����ڴ��Լ
	int i = txPerBlock * 8;
	while (i > 64) 
	{	
		tid = threadIdx.x;
		while(tid < (i >> 1))
		{
			value[tid] += value[tid + (i >> 1)];
			tid += txPerBlock;
		}  
		i >>= 1;
	}
	//�õ�ÿ�������Ӧ�ľ���64��ֵ
	tid = threadIdx.x;
	while(tid<64)
	{
  		blocks[tid + (blockIdx.x << 6)] = value[tid];	
		tid += txPerBlock;
	}
}
