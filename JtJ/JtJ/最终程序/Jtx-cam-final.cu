#include "hip/hip_runtime.h"
texture<float4, 1, hipReadModeElementType> tex_jc_cam;
texture<float4, 1, hipReadModeElementType> tex_x_cam;

__global__ void jtx_cam(int* result, int* cameraID, int *cameraCount)
{
	//分配共享内存
	__shared__ int value[txPerBlock * 8]; 
	//取出Jc矩阵元素以及对应的向量
	float4 x1 = tex1Dfetch(tex_x_cam, 0);
	float4 x2 = tex1Dfetch(tex_x_cam, 1);	
	int count = 0;
	int tid = threadIdx.x + (cameraID[blockIdx.x] >> 2);
	while (tid < cameraCount[blockIdx.x]>>2) 
	{
		//拾取绑定Jc矩阵的纹理内存
		float4 jc = tex1Dfetch(tex_jc_cam, tid);
		//载入共享内存
		int index = threadIdx.x<< 2;  
		value[index + 0] = jc.x;	value[index + 1] = jc.y;
		value[index + 2] = jc.z;	value[index + 3] = jc.w;
		//线程同步
		__syncthreads();
		index = threadIdx.x << 3;  
		if(threadIdx.x < (txPerBlock >> 1))
			result[threadIdx.x + count * (txPerBlock >> 1) + (cameraID[blockIdx.x] >> 3] 
				= value[index + 0] * x1.x + value[index + 1] * x1.y + value[index + 2] * x1.z + value[index + 3] * x1.w
			    + value[index + 4] * x2.x + value[index + 5] * x2.y + value[index + 6] * x2.z + value[index + 7] * x2.w;
		tid += txPerBlock;
		count ++ ;
	}
}