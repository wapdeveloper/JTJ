#include "hip/hip_runtime.h"
#include<stdio.h>
#include < hip/hip_runtime.h>
#include <d:/book.h>
#define N   1000
#define E    125


#define M   16
//#define M   256
#define K   2
#define threadsPerBlock  32

texture<int4, 1, hipReadModeElementType> tex_jtf_cam;
texture<int, 1, hipReadModeElementType> tex_f_cam;



__global__ void jtf_cam( int* blocks,int* cameraID ,int *cameraCount)
{
	////���乲���ڴ�
	__shared__ int value[threadsPerBlock*8]; 
	int rx = 0, ry = 0, rz = 0, rw = 0; 
	int tid =threadIdx.x+cameraID[blockIdx.x]/8;
	while (tid < cameraCount[blockIdx.x]/8) 
	{
		int4 jab= tex1Dfetch(tex_jtf_cam, (tid*2)+threadIdx.y);
		int err= tex1Dfetch(tex_f_cam, tid);
		rx += jab.x * err;		ry += jab.y * err;
        rz += jab.z * err;  	rw += jab.w * err;
		tid += blockDim.x;
	}

	int index=(threadIdx.x+(threadIdx.y*threadsPerBlock))<<2;  //5->32
	value[index+0]=rx;		value[index+1]=ry;
	value[index+2]=rz;		value[index+3]=rw;

	int i=8*threadsPerBlock;
	while (i > 8) 
	{	
		tid =threadIdx.x+(threadIdx.y<<7);
		while(tid < (i/4+(threadIdx.y<<7)))
		{
			//�����ڴ��Լ
			value[tid] += value[tid + i/4];
			tid += threadsPerBlock;
		}  
		i /= 2;
	}
	if(threadIdx.x < 4)
        blocks[threadIdx.x+(threadIdx.y<<2)+(blockIdx.x<<3)]=value[threadIdx.x+(threadIdx.y<<7)];
}






int main( void ) 
{
	int j[N], f[E], jtf[M],cameraID[K],cameraCount[K];
	int *dev_j,  *dev_f, *dev_jtf ,*dev_cameraID,*dev_cameraCount;

	//����GPU�Դ�
	HANDLE_ERROR( hipMalloc( (void**)&dev_j, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_f, E * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_jtf, M * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_cameraID, K * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_cameraCount, K * sizeof(int) ) );

	//��ȡ�ſɱȾ���
	const char *rpc1Filename = "D:\\in.txt";
	FILE* fid21 = fopen(rpc1Filename, "rt");
	for (int i = 0; i < N; i++)
	{
		fscanf(fid21, "%d ", &j[i]);
	}

	//��ȡ������
	rpc1Filename = "D:\\f.txt";
	fid21 = fopen(rpc1Filename, "rt");
	for (int i = 0; i < N; i++)
	{
		fscanf(fid21, "%d ", &f[i]);
	}

	cameraID[0]=0;
	cameraID[1]=256;

	cameraCount[0]=256;
	cameraCount[1]=1000;






	// �ſɱȾ��󿽱���GPU
	HANDLE_ERROR( hipMemcpy( dev_j, j, N * sizeof(int),hipMemcpyHostToDevice ) );	
	HANDLE_ERROR( hipMemcpy( dev_f, f, E * sizeof(int),hipMemcpyHostToDevice ) );	
	HANDLE_ERROR( hipMemcpy( dev_jtf, jtf, M * sizeof(int),	hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_cameraID, cameraID, K * sizeof(int),	hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_cameraCount, cameraCount, K * sizeof(int),	hipMemcpyHostToDevice ) );


	HANDLE_ERROR(hipBindTexture(0, tex_jtf_cam, dev_j, sizeof(int)*N));
	HANDLE_ERROR(hipBindTexture(0, tex_f_cam, dev_f, sizeof(int)*E));

	//�˺���
	dim3 grid(2), block(threadsPerBlock,2);
	jtf_cam<<<grid, block>>>( dev_jtf,dev_cameraID,dev_cameraCount );


	HANDLE_ERROR(hipUnbindTexture(tex_jtf_cam));
	HANDLE_ERROR(hipUnbindTexture(tex_f_cam));

	//��CPU������GPU
	//HANDLE_ERROR( hipMemcpy( j, dev_j, N * sizeof(int),hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy( f, dev_f, E * sizeof(int),hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy( jtf, dev_jtf, M * sizeof(int),hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy( cameraID, dev_cameraID, K * sizeof(int),hipMemcpyDeviceToHost ) );
	//HANDLE_ERROR( hipMemcpy(cameraCount, dev_cameraCount, K * sizeof(int),hipMemcpyDeviceToHost ) );


	// display the results
	for (int i=0; i<M; i++) 
	{
		printf( "%d   ", jtf[i] );
		if((i+1)%8==0)
			printf( "\n");
	}

	FILE * pFile;
	pFile = fopen("myfile.txt", "w");


	char x[M][10] = { "", };
	for (int i = 0; i<M; i++)
	{
		itoa(jtf[i], x[i], 10);
	}

	if (pFile != NULL)
	{
		for (size_t i = 0; i < M; i++)
		{
			fputs(x[i], pFile);
			fputs("\n", pFile);
		}
		fclose(pFile);
	}



	//free the memory allocated on the CPU
	//free(j);
	//free(f);
	//free(jtf);
	//free(cameraID);
	//free(cameraCount);

	// free the memory allocated on the GPU
	//HANDLE_ERROR( hipFree( dev_j ) );
	//HANDLE_ERROR( hipFree( dev_f ) );
	//HANDLE_ERROR( hipFree( dev_jtf ) );
	//HANDLE_ERROR( hipFree( dev_cameraID ) );
	//HANDLE_ERROR( hipFree( dev_cameraCount ) );



	return 0;
}






